#include "config.h"
#include <cstddef>
#include <hip/hip_runtime.h>
#include "CudaMemory.cuh"
#include "cuda_macro.h"
#include <cstring>

#define CUDA_EVENT 0x1201

/*! \brief Move the memory into device
 *
 * \return true if the memory is correctly flushed
 *
 */
bool CudaMemory::flush()
{
	if (hm != NULL && dm != NULL)
	{
		//! copy from host to device memory

		CUDA_SAFE_CALL(hipMemcpy(dm,hm,sz,hipMemcpyHostToDevice));		
	}
	
	return true;
}

/*! \brief Allocate a chunk of memory
 *
 * Allocate a chunk of memory
 *
 * \param sz size of the chunk of memory to allocate in byte
 *
 */
bool CudaMemory::allocate(size_t sz)
{
	//! Allocate the device memory
	if (dm == NULL)
	{CUDA_SAFE_CALL(hipMalloc(&dm,sz));}

	this->sz = sz;

	return true;
}

/*! \brief destroy a chunk of memory
 *
 * Destroy a chunk of memory
 *
 */
void CudaMemory::destroy()
{
	if (dm != NULL)
	{
		//! Release the allocated memory
		CUDA_SAFE_CALL(hipFree(dm));
		dm = NULL;
	}

	if (hm != NULL)
	{
		//! we invalidate hm
		CUDA_SAFE_CALL(hipHostFree(hm));
#ifdef SE_CLASS2
		//! remove hm
		check_delete(hm);
#endif
		hm = NULL;
	}
	
	sz = 0;
}

/*! \brief Allocate the host buffer
 *
 * Allocate the host buffer
 *
 */

void CudaMemory::allocate_host(size_t sz) const
{
	if (hm == NULL)
	{
		CUDA_SAFE_CALL(hipHostAlloc(&hm,sz,hipHostMallocMapped))
#ifdef SE_CLASS2
		//! add hm to the list of allocated memory
		check_new(hm,sz,CUDA_EVENT,0);
#endif
	}
}

/*! \brief copy the data from a pointer
 *
 * copy the data from a pointer
 *
 *	\param ptr
 *	\return true if success
 */
bool CudaMemory::copyFromPointer(const void * ptr)
{
	// check if we have a host buffer, if not allocate it

	allocate_host(sz);

	// get the device pointer

	void * dvp;
	CUDA_SAFE_CALL(hipHostGetDevicePointer(&dvp,hm,0));

	// memory copy

	memcpy(dvp,ptr,sz);

	return true;
}

/*! \brief copy from device to device
 *
 * copy a piece of memory from device to device
 *
 * \param CudaMemory from where to copy
 *
 * \return true is success
 */
bool CudaMemory::copyDeviceToDevice(const CudaMemory & m)
{
	//! The source buffer is too big to copy it

	if (m.sz > sz)
	{
		std::cerr << "Error " << __LINE__ << __FILE__ << ": source buffer is too big to copy";
		return false;
	}

	//! Copy the memory
	CUDA_SAFE_CALL(hipMemcpy(dm,m.dm,m.sz,hipMemcpyDeviceToDevice));

	return true;
}

/*! \brief copy from memory
 *
 * copy from memory
 *
 * \param m a memory interface
 *
 */
bool CudaMemory::copy(const memory & m)
{
	//! Here we try to cast memory into OpenFPMwdeviceCudaMemory
	const CudaMemory * ofpm = dynamic_cast<const CudaMemory *>(&m);

	//! if we fail we get the pointer and simply copy from the pointer

	if (ofpm == NULL)
	{
		// copy the memory from device to host and from host to device

		return copyFromPointer(m.getPointer());
	}
	else
	{
		// they are the same memory type, use cuda/thrust buffer copy

		return copyDeviceToDevice(*ofpm);
	}
}

/*! \brief Get the size of the allocated memory
 *
 * Get the size of the allocated memory
 *
 * \return the size of the allocated memory
 *
 */

size_t CudaMemory::size() const
{
	return sz;
}


/*! \brief Resize the allocated memory
 *
 * Resize the allocated memory, if request is smaller than the allocated memory
 * is not resized
 *
 * \param sz size
 * \return true if the resize operation complete correctly
 *
 */

bool CudaMemory::resize(size_t sz)
{
	// if the allocated memory is enough, do not resize
	if (sz <= size())
		return true;

	//! Allocate the device memory if not done yet

	if (size() == 0)
		return allocate(sz);

	//! Create a new buffer, if sz is bigger than the actual size
	void * thm;

	//! Create a new buffer, if sz is bigger than the actual size
	void * tdm;

	if (dm != NULL)
	{
		if (this->sz < sz)
			CUDA_SAFE_CALL(hipMalloc(&tdm,sz));

		//! copy from the old buffer to the new one

		CUDA_SAFE_CALL(hipMemcpy(tdm,dm,size(),hipMemcpyDeviceToDevice));
	}

	if (hm != NULL)
	{
		if (this->sz < sz)
			CUDA_SAFE_CALL(hipHostAlloc(&thm,sz,hipHostMallocMapped));

		//! copy from the old buffer to the new one

		CUDA_SAFE_CALL(hipMemcpy(thm,hm,size(),hipMemcpyHostToHost));
	}

	//! free the old buffer

	destroy();

	dm = tdm;
	hm = thm;

	//! change to the new buffer

	this->sz = sz;

	return true;
}

/*! \brief Return a readable pointer with your data
 *
 * \return a readable pointer with your data
 *
 */

void * CudaMemory::getPointer()
{
	// allocate an host memory if not allocated
	if (hm == NULL)
		allocate_host(sz);

	//! copy from device to host memory

	CUDA_SAFE_CALL(hipMemcpy(hm,dm,sz,hipMemcpyDeviceToHost));

	return hm;
}

/*! \brief Return a readable pointer with your data
 *
 * \return a readable pointer with your data
 *
 */

void CudaMemory::deviceToHost()
{
	// allocate an host memory if not allocated
	if (hm == NULL)
		allocate_host(sz);

	//! copy from device to host memory

	CUDA_SAFE_CALL(hipMemcpy(hm,dm,sz,hipMemcpyDeviceToHost));
}

/*! \brief Return a readable pointer with your data
 *
 * \return a readable pointer with your data
 *
 */

const void * CudaMemory::getPointer() const
{
	// allocate an host memory if not allocated
	if (hm == NULL)
		allocate_host(sz);

	//! copy from device to host memory

	CUDA_SAFE_CALL(hipMemcpy(hm,dm,sz,hipMemcpyDeviceToHost));

	return hm;
}

/*! \brief fill host and device memory with the selected byte
 *
 *
 */
void CudaMemory::fill(unsigned char c)
{
	CUDA_SAFE_CALL(hipMemset(dm,c,size()));
	memset(hm,c,size());
}

/*! \brief Return the CUDA device pointer
 *
 * \return CUDA device pointer
 *
 */
void * CudaMemory::getDevicePointer()
{
	// allocate an host memory if not allocated
	if (hm == NULL)
		allocate_host(sz);

	//! copy from device to host memory

	CUDA_SAFE_CALL(hipMemcpy(dm,hm,sz,hipMemcpyHostToDevice));

	return dm;
}

/*! \brief Return the CUDA device pointer (Do not copy to device)
 *
 * \return CUDA device pointer
 *
 */
void * CudaMemory::getDevicePointerNoCopy()
{
	return dm;
}

