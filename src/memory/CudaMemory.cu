#include <cstddef>
#include <hip/hip_runtime.h>
#include "CudaMemory.cuh"
#include "cuda_macro.h"

/*! \brief Allocate a chunk of memory
 *
 * Allocate a chunk of memory
 *
 * \param sz size of the chunk of memory to allocate in byte
 *
 */
bool CudaMemory::allocate(size_t sz)
{
	//! Allocate the device memory
	if (dm == NULL)
	{CUDA_SAFE_CALL(cudaAlloc(&dv,sz,cudaAlloc));}
}

/*! \brief destroy a chunk of memory
 *
 * Destroy a chunk of memory
 *
 */
void CudaMemory::destroy()
{
	CUDA_SAFE_CALL(cudaDestroy(dv));
}

/*! \brief copy the device memory to a pointer
 *
 * copy the device memory to a pointer
 *
 */
void CudaMemory::copyToPointer()
{
	// check if we have a host buffer, if not allocate it

	if (hm == NULL)
	{CUDA_SAFE_CALL(hipHostAlloc(&hm,sz*sizeof(T),hipHostMallocMapped))}

	// put on queue a copy from device to host

	t.call();

	// put on queue a memory copy from pointers
}

void CudaMemory::copyDeviceToDevice()
{
	// put on queue a copy from device to device

	t.call();
}

bool CudaMemory::copy(memory m)
{
	//! Here we try to cast memory into OpenFPMwdeviceCudaMemory
	CudaMemory * ofpm = dynamic_cast<CudaMemory>(m);

	//! if we fail we get the pointer and simply copy from the pointer

	if (ofpm == NULL)
	{
		// copy the memory from device to host and from host to device

		copyFromPointer(t);
	}
	else
	{
		// they are the same memory type, use cuda/thrust buffer copy

		copyDeviceToDevice();
	}
}

bool CudaMemory::copy(OpenFPMwdeviceCudaMemory m)
{
	// they are the same type of memory so copy from device to device

	copyDeviceToDevice();
}

size_t CudaMemory::size()
{
	dv->size();
}

bool CudaMemory::resize(size_t sz)
{
	//! Allocate the device memory
	if (dv == NULL)
	{dv = new boost::shared_ptr<void>(new thrust::device_vector<void>());}
	else
	{dv.get()->resize(sz);}
}
