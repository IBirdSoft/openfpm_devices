#include "config.h"
#include <cstddef>
#include <hip/hip_runtime.h>
#include "CudaMemory.cuh"
#include "cuda_macro.h"
#include <cstring>

/*! \brief Allocate a chunk of memory
 *
 * Allocate a chunk of memory
 *
 * \param sz size of the chunk of memory to allocate in byte
 *
 */
bool CudaMemory::allocate(size_t sz)
{
	//! Allocate the device memory
	if (dm == NULL)
	{CUDA_SAFE_CALL(hipMalloc(&dm,sz));}

	this->sz = sz;

	return true;
}

/*! \brief destroy a chunk of memory
 *
 * Destroy a chunk of memory
 *
 */
void CudaMemory::destroy()
{
	if (dm != NULL)
	{
		//! Release the allocated memory
		CUDA_SAFE_CALL(hipFree(dm));
		dm = NULL;
	}

	if (hm != NULL)
	{
		//! we invalidate hm
		CUDA_SAFE_CALL(hipHostFree(hm));
#ifdef SE_CLASS2
		//! remove hm
		check_delete(hm);
#endif
		hm = NULL;
	}
}

/*! \brief Allocate the host buffer
 *
 * Allocate the host buffer
 *
 */

void CudaMemory::allocate_host(size_t sz)
{
	if (hm == NULL)
	{
		CUDA_SAFE_CALL(hipHostAlloc(&hm,sz,hipHostMallocMapped))
#ifdef SE_CLASS2
		//! add hm to the list of allocated memory
		check_new(hm,sz);
#endif
	}
}

/*! \brief copy the data from a pointer
 *
 * copy the data from a pointer
 *
 *	\param ptr
 *	\return true if success
 */
bool CudaMemory::copyFromPointer(void * ptr)
{
	// check if we have a host buffer, if not allocate it

	allocate_host(sz);

	// get the device pointer

	void * dvp;
	CUDA_SAFE_CALL(hipHostGetDevicePointer(&dvp,hm,0));

	// memory copy

	memcpy(ptr,dvp,sz);

	return true;
}

/*! \brief copy from device to device
 *
 * copy a piece of memory from device to device
 *
 * \param CudaMemory from where to copy
 *
 * \return true is success
 */
bool CudaMemory::copyDeviceToDevice(CudaMemory & m)
{
	//! The source buffer is too big to copy it

	if (m.sz > sz)
	{
		std::cerr << "Error " << __LINE__ << __FILE__ << ": source buffer is too big to copy";
		return false;
	}

	//! Copy the memory
	CUDA_SAFE_CALL(hipMemcpy(m.dm,dm,m.sz,hipMemcpyDeviceToDevice));

	return true;
}

/*! \brief copy from memory
 *
 * copy from memory
 *
 * \param m a memory interface
 *
 */
bool CudaMemory::copy(memory & m)
{
	//! Here we try to cast memory into OpenFPMwdeviceCudaMemory
	CudaMemory * ofpm = dynamic_cast<CudaMemory *>(&m);

	//! if we fail we get the pointer and simply copy from the pointer

	if (ofpm == NULL)
	{
		// copy the memory from device to host and from host to device

		return copyFromPointer(m.getPointer());
	}
	else
	{
		// they are the same memory type, use cuda/thrust buffer copy

		return copyDeviceToDevice(*ofpm);
	}
}

/*! \brief Get the size of the allocated memory
 *
 * Get the size of the allocated memory
 *
 * \return the size of the allocated memory
 *
 */

size_t CudaMemory::size()
{
	return sz;
}

/*! \brief Resize the allocated memory
 *
 * Resize the allocated memory, if request is smaller than the allocated memory
 * is not resized
 *
 * \param sz size
 * \return true if the resize operation complete correctly
 *
 */

bool CudaMemory::resize(size_t sz)
{
	// if the allocated memory is enough, do not resize
	if (sz <= size())
		return true;

	//! Allocate the device memory if not done yet

	if (size() == 0)
		return allocate(sz);

	//! Create a new buffer, if sz is bigger than the actual size
	void * thm;

	//! Create a new buffer, if sz is bigger than the actual size
	void * tdm;

	if (dm != NULL)
	{
		if (this->sz < sz)
			CUDA_SAFE_CALL(hipMalloc(&tdm,sz));

		//! copy from the old buffer to the new one

		CUDA_SAFE_CALL(hipMemcpy(tdm,dm,size(),hipMemcpyDeviceToDevice));
	}

	if (hm != NULL)
	{
		if (this->sz < sz)
			CUDA_SAFE_CALL(hipHostAlloc(&thm,sz,hipHostMallocMapped));

		//! copy from the old buffer to the new one

		CUDA_SAFE_CALL(hipMemcpy(thm,hm,size(),hipMemcpyHostToHost));
	}

	//! free the old buffer

	destroy();

	dm = tdm;
	hm = thm;

	//! change to the new buffer

	this->sz = sz;

	return true;
}

/*! \brief Return a readable pointer with your data
 *
 * Return a readable pointer with your data
 *
 */

void * CudaMemory::getPointer()
{
	//| allocate an host memory if not allocated
	if (hm == NULL)
		allocate_host(sz);

	//! if the host buffer is synchronized with the device buffer return the host buffer

	if (is_hm_sync)
		return hm;

	//! copy from device to host memory

	CUDA_SAFE_CALL(hipMemcpy(hm,dm,sz,hipMemcpyDeviceToHost));

	return hm;
}
